#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "tools.cpp"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdio>
#include <chrono>
#include <cstring>
#include <cassert>
#include <thread>

#define N 1280
#define MATRIX_SIZE (N*N)
#define BLOCK_SIZE 16

#define timer_start (std::chrono::high_resolution_clock::now())
#if defined timer_start
    #define timer_elapsed(t) (std::chrono::duration_cast \
                                <std::chrono::nanoseconds> \
                                (std::chrono::high_resolution_clock \
                                        ::now() - t).count())
#endif

void simple_matrix_mul_cpu(float *matrix_a, float *matrix_b, float *matrix_c, uint64_t n) {
    float sum = 0.0;
    for (size_t i = 0; i < n; ++i) {
        for (size_t j = 0; j < n; ++j) {
            float sum = 0.0;
            for (size_t k = 0; k < n; ++k) {
                sum += matrix_a[i*n + k] * matrix_b[k*n + j];
            }
            matrix_c[i*n + j] = sum;
        }
    }
}

__global__ void simple_matrix_mul_gpu(float *matrix_a, float *matrix_b, float *matrix_c, uint64_t n) {
    uint64_t i = blockIdx.y * blockDim.y + threadIdx.y;
    uint64_t j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j < n) {
        float sum = 0.0;
        for (uint64_t k = 0; k < n; k++) {
            sum += matrix_a[i*n + k] * matrix_b[k*n + j];
        }
        matrix_c[i*n + j] = sum;
    }
}

__global__ void matrix_mul_gpu(float *matrix_a, float *matrix_b, float *matrix_c, uint64_t n) {
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin = n * BLOCK_SIZE * by;
    int aEnd   = aBegin + n - 1;
    int aStep  = BLOCK_SIZE;
    int bBegin = BLOCK_SIZE * bx;
    int bStep  = BLOCK_SIZE * n;
    float Csub = 0;

    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        As[ty][tx] = matrix_a[a + n * ty + tx];
        Bs[ty][tx] = matrix_b[b + n * ty + tx];
        __syncthreads();
        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Bs[k][tx];
        }
        __syncthreads();
    }

    int c = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    matrix_c[c + n * ty + tx] = Csub;
}

void HostVsDevice(uint8_t size, std::ostream &ost) {

    float *host_a, *host_b, *host_c, *buff_device_on_host;
    float *device_a, *device_b, *device_c;
    // alloc mem on host
    checkCudaErrors(hipHostMalloc(&buff_device_on_host, MATRIX_SIZE * sizeof(float)));
    checkCudaErrors(hipHostMalloc(&host_a, MATRIX_SIZE * sizeof(float)));
    checkCudaErrors(hipHostMalloc(&host_b, MATRIX_SIZE * sizeof(float)));
    checkCudaErrors(hipHostMalloc(&host_c, MATRIX_SIZE * sizeof(float)));

    // alloc mem on device
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&device_a), MATRIX_SIZE * sizeof(float)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&device_b), MATRIX_SIZE * sizeof(float)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&device_c), MATRIX_SIZE * sizeof(float)));

    // create cuda events to measure timing 
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // init host memory
    fill_random(host_a, MATRIX_SIZE);
    fill_random(host_b, MATRIX_SIZE);

    // copy arrays to device
    checkCudaErrors(hipMemcpy(device_a, host_a, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_b, host_b, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // exec calc on host
    auto timer = timer_start;
    simple_matrix_mul_cpu(host_a, host_b, host_c, N);
    auto elapsed = timer_elapsed(timer);
    ost << "Host calcs took: " << elapsed/1000000.0 << "ms" << std::endl;

    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // exec calc on device
    checkCudaErrors(hipEventRecord(start, 0));
    simple_matrix_mul_gpu <<<dimGrid, dimBlock>>> (device_a, device_b, device_c, N);
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);
    ost << "Device calcs took: " << msecTotal << "ms" << std::endl; 

    // copy calcs from device to host
    checkCudaErrors(hipMemcpy(buff_device_on_host, device_c, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost));
    if (compare_array(host_c, buff_device_on_host, MATRIX_SIZE))
        ost << "Computed matrixes from device and host are identical" << std::endl;
    else 
        ost << "Computed matrixes from device and host aren't identical" << std::endl;

    // set up cache config
    checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(matrix_mul_gpu), hipFuncCachePreferShared));

    checkCudaErrors(hipEventRecord(start, 0));
    matrix_mul_gpu <<<dimGrid, dimBlock>>> (device_a, device_b, device_c, N);
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));

    msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);
    ost << "Device calcs took: " << msecTotal << "ms" << std::endl;

    // copy calcs from device to host
    checkCudaErrors(hipMemcpy(buff_device_on_host, device_c, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost));
    if (compare_array(host_c, buff_device_on_host, MATRIX_SIZE))
        ost << "Computed matrixes from device and host are identical" << std::endl;
    else 
        ost << "Computed matrixes from device and host aren't identical" << std::endl;

    checkCudaErrors(hipFree(device_a));
    checkCudaErrors(hipFree(device_b));
    checkCudaErrors(hipFree(device_c));

    checkCudaErrors(hipHostFree(buff_device_on_host));
    checkCudaErrors(hipHostFree(host_a));
    checkCudaErrors(hipHostFree(host_b));
    checkCudaErrors(hipHostFree(host_c));
}

int main(int argc, char **argv) {
    int devID = findCudaDevice(argc, (const char **)argv);
    hipSetDevice(devID);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, devID);
    std::cout << "Device name: " << deviceProp.name << std::endl;
    std::cout << "Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
    std::cout << "Shared mem per multiprocessor: " << deviceProp.sharedMemPerMultiprocessor/1024.0 << " KB" << std::endl;
    std::cout << "L2 cache size: " << deviceProp.l2CacheSize/1024.0 << " KB" << std::endl;
    std::cout << "Const mem size: " << deviceProp.totalConstMem/1024.0 << " KB" << std::endl;
    std::cout << "Global mem size: " << deviceProp.totalGlobalMem/1073741824.0 << " GB" << std::endl;


    HostVsDevice(MATRIX_SIZE, std::cout);
    return 0;
}
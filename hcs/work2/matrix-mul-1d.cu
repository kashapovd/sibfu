#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "tools.cpp"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdio>
#include <chrono>
#include <cstring>
#include <cassert>
#include <thread>

#define N 160
#define MATRIX_SIZE (N*N)
#define BLOCK_SIZE 32
#define THREADS_PER_BLOCKS 1024

#define timer_start (std::chrono::high_resolution_clock::now())
#if defined timer_start
    #define timer_elapsed(t) (std::chrono::duration_cast \
                                <std::chrono::nanoseconds> \
                                (std::chrono::high_resolution_clock \
                                        ::now() - t).count())
#endif

void simple_matrix_mul_cpu(float *matrix_a, float *matrix_b, float *matrix_c, uint64_t n) {
    float sum = 0.0;
    for (size_t i = 0; i < n; ++i) {
        for (size_t j = 0; j < n; ++j) {
            float sum = 0.0;
            for (size_t k = 0; k < n; ++k) {
                sum += matrix_a[i*n + k] * matrix_b[k*n + j];
            }
            matrix_c[i*n + j] = sum;
        }
    }
}

__global__ void simple_matrix_mul_gpu(float *matrix_a, float *matrix_b, float *matrix_c, uint64_t n) {
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        uint64_t j = tid / N;
        uint64_t i = tid % N;
        float sum = 0.0;
        for (uint64_t k = 0; k < N; k++) {
            sum += matrix_a[i*N + k] * matrix_b[k*N + j];
        }
        matrix_c[i*N + j] = sum;
    }
}

void HostVsDevice(uint8_t size) {

    float *host_a, *host_b, *host_c, *buff_device_on_host;
    float *device_a, *device_b, *device_c;
    // alloc mem on host
    checkCudaErrors(hipHostMalloc(&buff_device_on_host, MATRIX_SIZE * sizeof(float)));
    checkCudaErrors(hipHostMalloc(&host_a, MATRIX_SIZE * sizeof(float)));
    checkCudaErrors(hipHostMalloc(&host_b, MATRIX_SIZE * sizeof(float)));
    checkCudaErrors(hipHostMalloc(&host_c, MATRIX_SIZE * sizeof(float)));

    // alloc mem on device
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&device_a), MATRIX_SIZE * sizeof(float)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&device_b), MATRIX_SIZE * sizeof(float)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&device_c), MATRIX_SIZE * sizeof(float)));

    // create cuda events to measure timing 
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // init host memory
    fill_random(host_a, MATRIX_SIZE);
    fill_random(host_b, MATRIX_SIZE);

    // copy arrays to device
    checkCudaErrors(hipMemcpy(device_a, host_a, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_b, host_b, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // exec calc on host
    auto timer = timer_start;
    simple_matrix_mul_cpu(host_a, host_b, host_c, N);
    auto elapsed = timer_elapsed(timer);
    std::cout << "Host calcs took: " << elapsed/1000000.0 << "ms" << std::endl;

    // exec calc on device
    checkCudaErrors(hipEventRecord(start, 0));
    simple_matrix_mul_gpu <<<BLOCK_SIZE, THREADS_PER_BLOCKS>>> (device_a, device_b, device_c, MATRIX_SIZE);
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);
    std::cout << "Device calcs took: " << msecTotal << "ms" << std::endl; 

    // copy calcs from device to host
    checkCudaErrors(hipMemcpy(buff_device_on_host, device_c, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost));
    if (compare_array(host_c, buff_device_on_host, MATRIX_SIZE))
        std::cout << "Computed matrixes from device and host are identical" << std::endl;
    else 
        std::cout << "Computed matrixes from device and host aren't identical" << std::endl;

    // set up cache config
    //checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(matrix_mul_gpu), hipFuncCachePreferShared));

    // checkCudaErrors(hipEventRecord(start, 0));
    // matrix_mul_gpu <<<dimGrid, dimBlock>>> (device_a, device_b, device_c, N);
    // checkCudaErrors(hipEventRecord(stop, 0));
    // checkCudaErrors(hipEventSynchronize(stop));

    // msecTotal = 0.0f;
    // hipEventElapsedTime(&msecTotal, start, stop);
    // std::cout << "Device calcs took: " << msecTotal << "ms" << std::endl;

    // // copy calcs from device to host
    // checkCudaErrors(hipMemcpy(buff_device_on_host, device_c, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost));
    // if (compare_array(host_c, buff_device_on_host, MATRIX_SIZE))
    //     std::cout << "Computed matrixes from device and host are identical" << std::endl;
    // else 
    //     std::cout << "Computed matrixes from device and host aren't identical" << std::endl;

    checkCudaErrors(hipFree(device_a));
    checkCudaErrors(hipFree(device_b));
    checkCudaErrors(hipFree(device_c));

    checkCudaErrors(hipHostFree(buff_device_on_host));
    checkCudaErrors(hipHostFree(host_a));
    checkCudaErrors(hipHostFree(host_b));
    checkCudaErrors(hipHostFree(host_c));
}

int main(int argc, char **argv) {
    int devID = findCudaDevice(argc, (const char **)argv);
    hipSetDevice(devID);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, devID);
    std::cout << "Device name: " << deviceProp.name << std::endl;
    std::cout << "Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
    std::cout << "Shared mem per multiprocessor: " << deviceProp.sharedMemPerMultiprocessor/1024.0 << " KB" << std::endl;
    std::cout << "L2 cache size: " << deviceProp.l2CacheSize/1024.0 << " KB" << std::endl;
    std::cout << "Const mem size: " << deviceProp.totalConstMem/1024.0 << " KB" << std::endl;
    std::cout << "Global mem size: " << deviceProp.totalGlobalMem/1073741824.0 << " GB" << std::endl;
    
    HostVsDevice(MATRIX_SIZE);
    return 0;
}
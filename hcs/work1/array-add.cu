#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "tools.cpp"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdio>
#include <chrono>
#include <cstring>
#include <cassert>


#define ARRAY_SIZE 100000
#define THREADS_PER_BLOCKS 1024

#define timer_start (std::chrono::high_resolution_clock::now())
#if defined timer_start
    #define timer_elapsed(t) (std::chrono::duration_cast \
                                <std::chrono::milliseconds> \
                                (std::chrono::high_resolution_clock \
                                        ::now() - t).count())
#endif

void vector_add_cpu(int32_t *host_a, int32_t *host_b, int32_t *host_c, uint64_t n) {
    uint64_t i;
    for (i = 0; i < n; ++i)
        host_c[i] = host_a[i] + host_b[i];
}


__global__ void vector_add_gpu(int32_t *device_a, int32_t *device_b, int32_t *device_c, uint64_t n) {
    int i =  blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        device_c[i] = device_a[i] + device_b[i];
}

int main() {

    int32_t *host_a, *host_b, *host_c;
    int32_t *device_a, *device_b, *device_c;

    // malloc mem on host
    host_a = (int32_t *)malloc(ARRAY_SIZE * sizeof(int32_t));
    host_b = (int32_t *)malloc(ARRAY_SIZE * sizeof(int32_t));
    host_c = (int32_t *)malloc(ARRAY_SIZE * sizeof(int32_t));

    // malloc mem on device
    auto r = hipMalloc(&device_a, ARRAY_SIZE * sizeof(int32_t));
    assert((r == hipError_t::hipSuccess));
    r = hipMalloc(&device_a, ARRAY_SIZE * sizeof(int32_t));
    assert((r == hipError_t::hipSuccess));
    r = hipMalloc(&device_a, ARRAY_SIZE * sizeof(int32_t));
    assert((r == hipError_t::hipSuccess));

    fill_random(host_a, ARRAY_SIZE);
    fill_random(host_b, ARRAY_SIZE);

    hipMemcpy(device_a, host_a, ARRAY_SIZE * sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, ARRAY_SIZE * sizeof(int32_t), hipMemcpyHostToDevice);

    auto timer = timer_start;
    vector_add_cpu(host_a, host_b, host_c, ARRAY_SIZE);
    auto elapsed = timer_elapsed(timer);
    std::cout << "vector_add_cpu_int: " << elapsed << " seconds" << std::endl;

    auto blocks = (ARRAY_SIZE + THREADS_PER_BLOCKS - 1) / THREADS_PER_BLOCKS;
    timer = timer_start;
    vector_add_gpu <<<blocks, THREADS_PER_BLOCKS>>> (device_a, device_b, device_c, ARRAY_SIZE);
    hipDeviceSynchronize();
    elapsed = timer_elapsed(timer);
    std::cout << "vector_add_gpu_int: " << elapsed << " seconds" << std::endl;

    auto buff_device_on_host = (int32_t *)malloc(ARRAY_SIZE * sizeof(int32_t));
    hipMemcpy(buff_device_on_host, device_c, ARRAY_SIZE * sizeof(int32_t), hipMemcpyDeviceToHost);
    if (compare_array(host_c, buff_device_on_host, ARRAY_SIZE))
        std::cout << "Computed arrays from device and host are identical" << std::endl;
    else 
        std::cout << "Computed arrays from device and host aren't identical" << std::endl;

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_b);

    free(host_a);
    free(host_b);
    free(host_c);

    return 0;
}
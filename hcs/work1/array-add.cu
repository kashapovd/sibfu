#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include "tools.cpp"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdio>
#include <chrono>
#include <cstring>
#include <cassert>

#define INIT_ARRAY_SIZE 100
#define THREADS_PER_BLOCKS 1024

#define timer_start (std::chrono::high_resolution_clock::now())
#if defined timer_start
    #define timer_elapsed(t) (std::chrono::duration_cast \
                                <std::chrono::nanoseconds> \
                                (std::chrono::high_resolution_clock \
                                        ::now() - t).count())
#endif

void vector_add_cpu(int32_t *host_a, int32_t *host_b, int32_t *host_c, uint64_t n) {
    uint64_t i;
    for (i = 0; i < n; ++i)
        host_c[i] = host_a[i] + host_b[i];
}

__global__ void vector_add_gpu(int32_t *device_a, int32_t *device_b, int32_t *device_c, uint64_t n) {
    int i =  blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        device_c[i] = device_a[i] + device_b[i];
}

void HostVsDevice(size_t size, std::ostream &ost) {
    auto device_blocks = (size + THREADS_PER_BLOCKS - 1) / THREADS_PER_BLOCKS;

    int32_t *host_a, *host_b, *host_c, *buff_device_on_host;
    int32_t *device_a, *device_b, *device_c;

    // alloc mem on host
    checkCudaErrors(hipHostMalloc(&buff_device_on_host, size * sizeof(int32_t)));
    checkCudaErrors(hipHostMalloc(&host_a, size * sizeof(int32_t)));
    checkCudaErrors(hipHostMalloc(&host_b, size * sizeof(int32_t)));
    checkCudaErrors(hipHostMalloc(&host_c, size * sizeof(int32_t)));

    // alloc mem on device
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&device_a), size * sizeof(int32_t)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&device_b), size * sizeof(int32_t)));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&device_c), size * sizeof(int32_t)));

    // create cuda events to measure timing 
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // init host memory
    fill_random(host_a, size);
    fill_random(host_b, size);

    checkCudaErrors(hipMemcpy(device_a, host_a, size * sizeof(int32_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_b, host_b, size * sizeof(int32_t), hipMemcpyHostToDevice));

    // exec calc on host
    auto timer = timer_start;
    vector_add_cpu(host_a, host_b, host_c, size);
    auto elapsed = timer_elapsed(timer);
    
    ost << "Host calcs took: " << float(elapsed/1000000.0) << "ms" << std::endl;

    checkCudaErrors(hipProfilerStart());
    // exec calc on device
    checkCudaErrors(hipEventRecord(start, 0));
    vector_add_gpu <<<device_blocks, THREADS_PER_BLOCKS>>> (device_a, device_b, device_c, size);
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipProfilerStop());

    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);
    ost << "Device calcs took: " << msecTotal << "ms" << std::endl; 

    checkCudaErrors(hipMemcpy(buff_device_on_host, device_c, size * sizeof(int32_t), hipMemcpyDeviceToHost));

    if (compare_array(host_c, buff_device_on_host, size))
        ost << "Computed arrays from device and host are identical" << std::endl;
    else 
        ost << "Computed arrays from device and host aren't identical" << std::endl;

    checkCudaErrors(hipFree(device_a));
    checkCudaErrors(hipFree(device_b));
    checkCudaErrors(hipFree(device_c));

    checkCudaErrors(hipHostFree(buff_device_on_host));
    checkCudaErrors(hipHostFree(host_a));
    checkCudaErrors(hipHostFree(host_b));
    checkCudaErrors(hipHostFree(host_c));

}

int main(int argc, char **argv) {
    int devID = findCudaDevice(argc, (const char **)argv);
    
    for (int i = 1; i <= 5; i++) {
        std::cout << "Test №" << i << " === Array size: " << INIT_ARRAY_SIZE*std::pow(10, i) << std::endl; 
        HostVsDevice(INIT_ARRAY_SIZE*std::pow(10, i), std::cout);
        std::cout << std::endl;
    }
    return 0;
}
